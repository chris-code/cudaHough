#include "hip/hip_runtime.h"
#include "houghTransform.h"

// this method transforms a rgb-color image to an grayvalue image
CImg<double> RGBToGrayValueImage(const CImg<double> &image) {
	// initialize the gray value image
	CImg<double> grayImg(image.width(), image.height(), 1, 1);

	// iterate over the image
	for (int i = 0; i < image.width(); i++)
		for (int j = 0; j < image.height(); j++) {
			// The gray value is calculated by the following formula: 0.21 R + 0.72 G + 0.07 B
			grayImg(i, j, 0, 0) = 0.21 * image(i, j, 0, 0) + 0.72 * image(i, j, 0, 1) + 0.07 * image(i, j, 0, 2);
		}

	return grayImg;
}

#define assertCheck(arg) { errorCheck(arg, __FILE__, __LINE__); }
void errorCheck(hipError_t returnCode, char *file, long line) {
	if (returnCode != hipSuccess) {
		std::cerr << hipGetErrorString(returnCode) << " occured at " << line << " in file " << file << std::endl;
		exit(EXIT_FAILURE);
	}
}

double * cImgToGPU(CImg<double> image) {
	double *gpuImage;
	assertCheck( hipMalloc(&gpuImage, image.height() * image.width() * sizeof(double)) );
	assertCheck(
			hipMemcpy(gpuImage, image.data(), image.height() * image.width() * sizeof(double), hipMemcpyHostToDevice));
	return gpuImage;
}

CImg<double> gpuToCImg(double *image, long width, long height) {
	double *cpuData = new double[width * height];
	assertCheck( hipMemcpy(&cpuData, image, width * height * sizeof(double), hipMemcpyDeviceToHost) );
	CImg<double> cpuImg(cpuData, width, height);
	delete[] cpuData;
	return cpuImg;
}

__global__ void convolve(double *result, double *image, long imgWidth, long imgHeight, double *filter, long filWidth,
		long filHeight, long filAnchorX, long filAnchorY) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < imgWidth && y < imgHeight) {
		double value = 0;

		for (long filX = 0; filX < filWidth; ++filX) {
			long posImgX = ((x - filAnchorX + filX) + imgWidth) % imgWidth;
			for (long filY = 0; filY < filHeight; ++filY) {
				long posImgY = ((y - filAnchorY + filY) + imgHeight) % imgHeight;

				value += image[posImgY * imgWidth + posImgX] * filter[filY * filWidth + filX];
			}
		}

		image[y * imgWidth + x] = value;
	}
}

__global__ void computeGradientStrengthGPU(double *gradientStrength, double *gradientX, double *gradientY, long width,
		long height) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		long index = y * width + x;
		gradientStrength[index] = sqrt(pow(gradientX[index], 2) + pow(gradientY[index], 2));
	}
}

double * computeGradientStrength(double *grayValueImage, long width, long height) {
	double cpuSobelX[] = { 1, 0, -1, 2, 0, -2, 1, 0, -1 };
	double cpuSobelY[] = { 1, 2, 1, 0, 0, 0, -1, -2, -1 };
	double *sobelX;
	double *sobelY;
	assertCheck(hipMalloc(&sobelX, 9 * sizeof(double)));
	assertCheck(hipMalloc(&sobelY, 9 * sizeof(double)));
	assertCheck(hipMemcpy(sobelX, cpuSobelX, 9 * sizeof(double), hipMemcpyHostToDevice));
	assertCheck(hipMemcpy(sobelY, cpuSobelY, 9 * sizeof(double), hipMemcpyHostToDevice));

	double *gradientX;
	double *gradientY;
	assertCheck(hipMalloc(&gradientX, width * height * sizeof(double)));
	assertCheck(hipMalloc(&gradientY, width * height * sizeof(double)));

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
	convolve<<<blocks, threads>>>(gradientX, grayValueImage, width, height, sobelX, 3, 3, 1, 1);
	convolve<<<blocks, threads>>>(gradientY, grayValueImage, width, height, sobelY, 3, 3, 1, 1);

	double *gradientStrength;
	assertCheck(hipMalloc(&gradientStrength, width * height * sizeof(double)));
	computeGradientStrengthGPU<<<blocks, threads>>>(gradientStrength, gradientX, gradientY, width, height);

	hipFree(sobelX);
	hipFree(sobelY);
	hipFree(gradientX);
	hipFree(gradientY);

	return gradientStrength;
}

CImg<bool> cudaHough::preprocess(CImg<double> image) {
	CImg<double> cpuGrayValueImage = RGBToGrayValueImage(image);
	double *grayValueImage = cImgToGPU(cpuGrayValueImage);

	double *gradientStrengthImage = computeGradientStrength(grayValueImage, image.width(), image.height());

	CImg<double> CPUgradientStrenthImage = gpuToCImg(gradientStrengthImage, image.width(), image.height());
	CImgDisplay d(CPUgradientStrenthImage, "foo", 1);
	while(!d.is_closed()) {
		d.wait();
	}

	hipFree(grayValueImage);
}

CImg<long> cudaHough::transform(CImg<bool> binaryImage) {
//	TODO
}

std::vector<std::pair<double, double> > cudaHough::extractMostLikelyLines(CImg<long> accumulatorArray,
		long linesToExtract) {

}
