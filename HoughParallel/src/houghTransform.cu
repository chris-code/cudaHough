#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include "houghTransform.h"

// this method transforms a rgb-color image to an grayvalue image
template<typename T>
CImg<T> RGBToGrayValueImage(const CImg<T> &image) {
	// initialize the gray value image
	CImg<T> grayImg(image.width(), image.height(), 1, 1);

	// iterate over the image
	for (long i = 0; i < image.width(); ++i)
		for (long j = 0; j < image.height(); ++j) {
			// The gray value is calculated by the following formula: 0.21 R + 0.72 G + 0.07 B
			grayImg(i, j, 0, 0) = 0.21 * image(i, j, 0, 0) + 0.72 * image(i, j, 0, 1) + 0.07 * image(i, j, 0, 2);
		}

	return grayImg;
}

#define assertCheck(arg) { errorCheck((arg), __FILE__, __LINE__); }
void errorCheck(const hipError_t returnCode, const char *file, const long line) {
	if (returnCode != hipSuccess) {
		std::cerr << hipGetErrorString(returnCode) << " occurred at " << line << " in file " << file << std::endl;
		exit(EXIT_FAILURE);
	}
}

template<typename T>
T * cImgToGPU(CImg<T> &image) {
	T *gpuImage;
	assertCheck(hipMalloc(&gpuImage, image.height() * image.width() * sizeof(T)));
	assertCheck(hipMemcpy(gpuImage, image.data(), image.height() * image.width() * sizeof(T), hipMemcpyHostToDevice));
	return gpuImage;
}

template<typename imgT>
CImg<imgT> gpuToCImg(imgT *image, long width, long height, bool freeMemory) {
	imgT *cpuData = (imgT*) malloc(width * height * sizeof(imgT));
	assertCheck(hipMemcpy(cpuData, image, width * height * sizeof(imgT), hipMemcpyDeviceToHost));
	if (freeMemory)
		assertCheck(hipFree(image));
	CImg<imgT> cpuImg(cpuData, width, height);
	free(cpuData);
	return cpuImg;
}

template<typename T>
__global__ void convolve(T *result, T *image, long imgWidth, long imgHeight, T *filter, long filWidth, long filHeight,
	long filAnchorX, long filAnchorY) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < imgWidth && y < imgHeight) {
		T value = 0;

		for (long filX = 0; filX < filWidth; ++filX) {
			long posImgX = ((x - filAnchorX + filX) + imgWidth) % imgWidth;
			for (long filY = 0; filY < filHeight; ++filY) {
				long posImgY = ((y - filAnchorY + filY) + imgHeight) % imgHeight;

				value += image[posImgY * imgWidth + posImgX] * filter[filY * filWidth + filX];
			}
		}

		result[y * imgWidth + x] = value;
	}
}

//	Note that this kernel takes sigma^2 as an argument.
//	TODO this kernel could actually just take one dimension instead of width and height. See if this would run faster.
template<typename imgT>
__global__ void generateGauss(imgT *result, long width, long height, imgT sigma2, imgT normalizationTerm) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;
	long index = y * width + x;
	if (x < width && y < height) {
		imgT coordX = x - width / 2.0;
		imgT coordY = y - height / 2.0;

		imgT value = pow(coordX, imgT(2.0)) + pow(coordY, imgT(2.0));
		value /= 2 * sigma2;
		value = exp(-value);
		result[index] = value * normalizationTerm;
	}
}

template<typename imgT>
imgT * gaussBlurr(imgT *image, long width, long height, imgT sigma) {
	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);

//	Generate gaussian
	long filterWidth = 2 * sigma + 1; // 2-sigma rule, catch 95% of all values
	long filterHeight = 2 * sigma + 1; // (but make it odd so that a center exists)
//	TODO we get about 95% of all values. The gauss filter would normally sum to 1 since the gauss curve integrates to
//	1. But we lose 5% of it, so it doesn't exactly sum to one. Modify the normalization term to compensate for this.
	imgT normalizationTerm = 1.0 / (2.0 * M_PI * pow(sigma, 2.0));
	imgT *gauss;
	assertCheck(hipMalloc(&gauss, filterWidth * filterHeight * sizeof(imgT)));
	generateGauss<imgT> <<<blocks, threads>>>(gauss, filterWidth, filterHeight, imgT(pow(sigma, 2.0)),
		imgT(normalizationTerm));
	assertCheck(hipGetLastError());

//	Blurr the image with it
	imgT *result;
	assertCheck(hipMalloc(&result, width * height * sizeof(imgT)));
	convolve<imgT> <<<blocks, threads>>>(result, image, width, height, gauss, filterWidth, filterHeight,
		filterWidth / 2, filterHeight / 2);
	assertCheck(hipGetLastError());
	assertCheck(hipFree(gauss));

	return result;
}

template<typename T>
__global__ void computeGradientStrengthGPU(T *gradientStrength, T *gradientX, T *gradientY, long width, long height) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		long index = y * width + x;
		gradientStrength[index] = sqrt(pow(gradientX[index], 2) + pow(gradientY[index], 2));
	}
}

template<typename T>
T * computeGradientStrength(T *grayValueImage, long width, long height) {
	T cpuSobelX[] = {1, 0, -1, 2, 0, -2, 1, 0, -1};
	T cpuSobelY[] = {1, 2, 1, 0, 0, 0, -1, -2, -1};
	T *sobelX;
	T *sobelY;
	assertCheck(hipMalloc(&sobelX, 9 * sizeof(T)));
	assertCheck(hipMalloc(&sobelY, 9 * sizeof(T)));
	assertCheck(hipMemcpy(sobelX, cpuSobelX, 9 * sizeof(T), hipMemcpyHostToDevice));
	assertCheck(hipMemcpy(sobelY, cpuSobelY, 9 * sizeof(T), hipMemcpyHostToDevice));

	T *gradientX;
	T *gradientY;
	assertCheck(hipMalloc(&gradientX, width * height * sizeof(T)));
	assertCheck(hipMalloc(&gradientY, width * height * sizeof(T)));

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
	convolve<T> <<<blocks, threads>>>(gradientX, grayValueImage, width, height, sobelX, 3, 3, 1, 1);
	assertCheck(hipGetLastError());
	convolve<T> <<<blocks, threads>>>(gradientY, grayValueImage, width, height, sobelY, 3, 3, 1, 1);
	assertCheck(hipGetLastError());

	T *gradientStrength;
	assertCheck(hipMalloc(&gradientStrength, width * height * sizeof(T)));
	computeGradientStrengthGPU<T> <<<blocks, threads>>>(gradientStrength, gradientX, gradientY, width, height);
	assertCheck(hipGetLastError());

	assertCheck(hipFree(sobelX));
	assertCheck(hipFree(sobelY));
	assertCheck(hipFree(gradientX));
	assertCheck(hipFree(gradientY));

	return gradientStrength;
}

template<typename T>
__global__ void binarizeGPU(bool *result, T *image, long width, long height, T threshold) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		long index = y * width + x;
		if (image[index] > threshold)
			result[index] = 1;
		else
			result[index] = 0;
	}
}

template<typename T>
bool * binarize(T *image, long width, long height, T relativeThreshold) {
	thrust::device_ptr<T> imageThrustPointer(image);
	T min = (thrust::min_element(imageThrustPointer, imageThrustPointer + (width * height)))[0];
	T max = (thrust::max_element(imageThrustPointer, imageThrustPointer + (width * height)))[0];
	T absoluteThreshold = (max - min) * relativeThreshold + min;

	bool *binaryImage;
	assertCheck(hipMalloc(&binaryImage, width * height * sizeof(bool)));

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
	binarizeGPU<T> <<<blocks, threads>>>(binaryImage, image, width, height, absoluteThreshold);
	assertCheck(hipGetLastError());

	return binaryImage;
}

template<typename imgT>
bool * cudaHough::preprocess(CImg<imgT> &image, imgT relativeThreshold, imgT sigma) {
	imgT *grayValueImage = cImgToGPU<imgT>(image);
	imgT *blurredImage = gaussBlurr<imgT>(grayValueImage, image.width(), image.height(), sigma);
	imgT *gradientStrengthImage = computeGradientStrength<imgT>(blurredImage, image.width(), image.height());
	bool *binaryImage = binarize<imgT>(gradientStrengthImage, image.width(), image.height(), relativeThreshold);

	assertCheck(hipFree(grayValueImage));
	assertCheck(hipFree(blurredImage));
	assertCheck(hipFree(gradientStrengthImage));

	return binaryImage;
}

template<typename accuT, typename Tparam>
__global__ void computeAccumulatorArrayGPU(bool *binaryImage, long width, long height, long borderExclude,
	accuT *accumulatorArray, Tparam minTheta, Tparam maxTheta, Tparam thetaStepSize, Tparam stepsPerRadian,
	Tparam minR, Tparam stepsPerPixel, long dimTheta) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;
//	TODO calculate x and y by directly taking into account border exclude, instead of checking it afterwards
	if (x >= borderExclude && y >= borderExclude && x < width - borderExclude && y < height - borderExclude) {
		if (binaryImage[y * width + x] == 1) {
			for (Tparam theta = minTheta; theta < maxTheta; theta += thetaStepSize) {
				Tparam r = x * cos(theta) + y * sin(theta);

				long thetaIdx = long((theta - minTheta) * stepsPerRadian);
				long rIdx = long((r - minR) * stepsPerPixel);
				accumulatorArray[rIdx * dimTheta + thetaIdx] += 1;
			}
		}
	}
}

template<typename accuT, typename paramT>
accuT * cudaHough::transform(bool *binaryImage, long width, long height, HoughParameterSet<paramT> &hps) {
	long dimTheta = hps.getDimTheta();
	long dimR = hps.getDimR();
	long borderExclude = 5;

	accuT *accumulatorArray;
	assertCheck(hipMalloc(&accumulatorArray, dimTheta * dimR * sizeof(accuT)));
	hipMemset(accumulatorArray, 0, dimTheta * dimR * sizeof(accuT));

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
	computeAccumulatorArrayGPU<accuT, paramT> <<<blocks, threads>>>(binaryImage, width, height, borderExclude,
		accumulatorArray, hps.minTheta, hps.maxTheta, hps.getThetaStepSize(), hps.stepsPerRadian, hps.minR,
		hps.stepsPerPixel, dimTheta);
	assertCheck(hipGetLastError());

	return accumulatorArray;
}

template<typename T>
__global__ void isolateLocalMaximaGPU(T *accumulatorArray, T *localMaxima, long width, long height,
	long excludeRadius) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		long index = y * width + x;

		for (long offsetX = -excludeRadius; offsetX <= excludeRadius; ++offsetX) {
			long posX = ((x + offsetX) + width) % width;

			for (long offsetY = -excludeRadius; offsetY <= excludeRadius; ++offsetY) {
				long posY = ((y + offsetY) + height) % height;
				long offsetIndex = ((posY * width) + posX);

				if (accumulatorArray[offsetIndex] >= accumulatorArray[index] && offsetIndex != index) {
					localMaxima[index] = -1;
					return;
				}
			}
		}

		localMaxima[index] = accumulatorArray[index];
	}
}

template<typename T>
T * isolateLocalMaxima(T *accumulatorArray, long width, long height, long excludeRadius) {
	T *localMaxima;
	assertCheck(hipMalloc(&localMaxima, width * height * sizeof(T)));

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
	isolateLocalMaximaGPU<T> <<<blocks, threads>>>(accumulatorArray, localMaxima, width, height, excludeRadius);
	assertCheck(hipGetLastError());

	return localMaxima;
}

template<typename T>
thrust::device_vector<long> getSortedIndices(T *maxima, long width, long height) {
	thrust::device_vector<long> indices(width * height);
	thrust::sequence(indices.begin(), indices.end());

	thrust::device_ptr<T> maximaThrust(maxima);
	thrust::sort_by_key(maximaThrust, maximaThrust + width * height, indices.begin(), thrust::greater<long>());

	return indices; // TODO don't return thrust::device_vector
}

template<typename accuT, typename paramT>
std::vector<std::pair<paramT, paramT> > cudaHough::extractStrongestLines(accuT *accumulatorArray, long linesToExtract,
	long excludeRadius, HoughParameterSet<paramT> &hps) {
	accuT *localMaxima = isolateLocalMaxima(accumulatorArray, hps.getDimTheta(), hps.getDimR(), excludeRadius);
	thrust::device_vector<long> sortedIndices = getSortedIndices<accuT>(localMaxima, hps.getDimTheta(), hps.getDimR());
	thrust::host_vector<long> cpuSortedIndices(linesToExtract);
	thrust::copy(sortedIndices.begin(), sortedIndices.begin() + linesToExtract, cpuSortedIndices.begin());

	std::vector<std::pair<paramT, paramT> > bestLines;
	for (long i = 0; i < linesToExtract; i++) {
		long x = cpuSortedIndices[i] % hps.getDimTheta();
		long y = cpuSortedIndices[i] / hps.getDimTheta();

		double theta = hps.minTheta + hps.getThetaStepSize() * x;
		double r = hps.minR + hps.getRstepSize() * y;

		bestLines.push_back(std::make_pair<double, double>(theta, r));
	}
	assertCheck(hipFree(localMaxima));

	return bestLines;
}

template<typename imgT, typename accuT, typename paramT>
std::vector<std::pair<paramT, paramT> > cudaHough::extractStrongestLines(CImg<imgT> &image,
	HoughParameterSet<paramT> &hps, imgT binarizationThreshold, imgT sigma, long linesToExtract, long excludeRadius) {
	bool *binaryImage = preprocess<imgT>(image, binarizationThreshold, sigma);
	accuT *accumulatorArray = transform<accuT, paramT>(binaryImage, image.width(), image.height(), hps);
	std::vector<std::pair<paramT, paramT> > strongestLines = extractStrongestLines<accuT, paramT>(accumulatorArray,
		linesToExtract, excludeRadius, hps);

	assertCheck(hipFree(binaryImage));
	assertCheck(hipFree(accumulatorArray));

	return strongestLines;
}

// Instantiate template methods so they are available to the compiler
template CImg<bool> gpuToCImg(bool *image, long width, long height, bool freeMemory);
template CImg<long> gpuToCImg(long *image, long width, long height, bool freeMemory);
template CImg<float> gpuToCImg(float *image, long width, long height, bool freeMemory);
template CImg<double> gpuToCImg(double *image, long width, long height, bool freeMemory);
template bool * cudaHough::preprocess<float>(CImg<float> &image, float threshold, float sigma);
template bool * cudaHough::preprocess<double>(CImg<double> &image, double threshold, double sigma);
template long * cudaHough::transform(bool *binaryImage, long width, long height, HoughParameterSet<float> &hps);
template long * cudaHough::transform(bool *binaryImage, long width, long height, HoughParameterSet<double> &hps);
template std::vector<std::pair<float, float> > cudaHough::extractStrongestLines(long *accumulatorArray,
	long linesToExtract, long excludeRadius, HoughParameterSet<float> &hps);
template std::vector<std::pair<double, double> > cudaHough::extractStrongestLines(long *accumulatorArray,
	long linesToExtract, long excludeRadius, HoughParameterSet<double> &hps);
template std::vector<std::pair<float, float> > cudaHough::extractStrongestLines<float, long, float>(
	CImg<float> &image, HoughParameterSet<float> &hps, float threshold, float sigma, long linesToExtract,
	long excludeRadius);
template std::vector<std::pair<double, double> > cudaHough::extractStrongestLines<double, long, double>(
	CImg<double> &image, HoughParameterSet<double> &hps, double threshold, double sigma, long linesToExtract,
	long excludeRadius);
