#include "hip/hip_runtime.h"
#include "houghTransform.h"

// this method transforms a rgb-color image to an grayvalue image
CImg<double> RGBToGrayValueImage(const CImg<double> &image) {
	// initialize the gray value image
	CImg<double> grayImg(image.width(), image.height(), 1, 1);

	// iterate over the image
	for (int i = 0; i < image.width(); i++)
		for (int j = 0; j < image.height(); j++) {
			// The gray value is calculated by the following formula: 0.21 R + 0.72 G + 0.07 B
			grayImg(i, j, 0, 0) = 0.21 * image(i, j, 0, 0) + 0.72 * image(i, j, 0, 1) + 0.07 * image(i, j, 0, 2);
		}

	return grayImg;
}

#define assertCheck(arg) { errorCheck(arg, __FILE__, __LINE__); }
void errorCheck(hipError_t returnCode, char *file, long line) {
	if (returnCode != hipSuccess) {
		std::cerr << hipGetErrorString(returnCode) << " occured at " << line << " in file " << file << std::endl;
		exit(EXIT_FAILURE);
	}
}

double * cImgToGPU(CImg<double> image) {
	double *gpuImage;
	assertCheck(hipMalloc(&gpuImage, image.height() * image.width() * sizeof(double)));
	assertCheck(
			hipMemcpy(gpuImage, image.data(), image.height() * image.width() * sizeof(double), hipMemcpyHostToDevice));
	return gpuImage;
}

__global__ void convolve(double *result, double *image, long imgWidth, long imgHeight, double *filter, long filWidth,
		long filHeight, long filAnchorX, long filAnchorY) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < imgWidth && y < imgHeight) {
		double value = 0;

		for (long filX = 0; filX < filWidth; ++filX) {
			long posImgX = ((x - filAnchorX + filX) + imgWidth) % imgWidth;
			for (long filY = 0; filY < filHeight; ++filY) {
				long posImgY = ((y - filAnchorY + filY) + imgHeight) % imgHeight;

				value += image[posImgY * imgWidth + posImgX] * filter[filY * filWidth + filX];
			}
		}

		image[y * imgWidth + x] = value;
	}
}

double * computeGradientStrength(double *grayValueImage, long width, long height) {
	double cpuSobelX[] = { 1, 0, -1, 2, 0, -2, 1, 0, -1 };
	double cpuSobelY[] = { 1, 2, 1, 0, 0, 0, -1, -2, -1 };
	double *sobelX;
	double *sobelY;
	assertCheck(hipMalloc(&sobelX, 9 * sizeof(double)));
	assertCheck(hipMalloc(&sobelY, 9 * sizeof(double)));
	assertCheck(hipMemcpy(sobelX, cpuSobelX, 9 * sizeof(double), hipMemcpyHostToDevice));
	assertCheck(hipMemcpy(sobelY, cpuSobelY, 9 * sizeof(double), hipMemcpyHostToDevice));

	double *gradientX;
	double *gradientY;
	assertCheck(hipMalloc(&gradientX, width * height * sizeof(double)));
	assertCheck(hipMalloc(&gradientY, width * height * sizeof(double)));

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
	convolve<<<blocks, threads>>>(gradientX, grayValueImage, width, height, sobelX, 3, 3, 1, 1);
	convolve<<<blocks, threads>>>(gradientY, grayValueImage, width, height, sobelY, 3, 3, 1, 1);

	double *gradientStrength;
	assertCheck(hipMalloc(&gradientStrength, width * height * sizeof(double)));
//	TODO calc. gradient strength

	hipFree(sobelX);
	hipFree(sobelY);
	hipFree(gradientX);
	hipFree(gradientY);

	return gradientStrength;
}

CImg<bool> cudaHough::preprocess(CImg<double> image) {
	CImg<double> cpuGrayValueImage = RGBToGrayValueImage(image);
	double *grayValueImage = cImgToGPU(cpuGrayValueImage);

	double *gradientStrengthImage = computeGradientStrength(grayValueImage, image.width(), image.height());

	hipFree(grayValueImage);
}

CImg<long> cudaHough::transform(CImg<bool> binaryImage) {
//	TODO
}

std::vector<std::pair<double, double> > cudaHough::extractMostLikelyLines(CImg<long> accumulatorArray,
		long linesToExtract) {

}
