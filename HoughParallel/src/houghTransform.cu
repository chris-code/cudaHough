#include "hip/hip_runtime.h"
#include "houghTransform.h"

// this method transforms a rgb-color image to an grayvalue image
template<typename T>
CImg<T> RGBToGrayValueImage(const CImg<T> &image) {
	// initialize the gray value image
	CImg<T> grayImg(image.width(), image.height(), 1, 1);

	// iterate over the image
	for (long i = 0; i < image.width(); ++i)
		for (long j = 0; j < image.height(); ++j) {
			// The gray value is calculated by the following formula: 0.21 R + 0.72 G + 0.07 B
			grayImg(i, j, 0, 0) = 0.21 * image(i, j, 0, 0) + 0.72 * image(i, j, 0, 1) + 0.07 * image(i, j, 0, 2);
		}

	return grayImg;
}

#define assertCheck(arg) { errorCheck((arg), __FILE__, __LINE__); }
void errorCheck(const hipError_t returnCode, const char *file, const long line) {
	if (returnCode != hipSuccess) {
		std::cerr << hipGetErrorString(returnCode) << " occured at " << line << " in file " << file << std::endl;
		exit(EXIT_FAILURE);
	}
}

template<typename T>
T * cImgToGPU(CImg<T> &image) {
	T *gpuImage;
	assertCheck(hipMalloc(&gpuImage, image.height() * image.width() * sizeof(T)));
	assertCheck(hipMemcpy(gpuImage, image.data(), image.height() * image.width() * sizeof(T), hipMemcpyHostToDevice));
	return gpuImage;
}

template<typename T>
CImg<T> gpuToCImg(T *image, long width, long height) {
	T *cpuData = (T*) malloc(width * height * sizeof(T));
	assertCheck(hipMemcpy(cpuData, image, width * height * sizeof(T), hipMemcpyDeviceToHost));
	assertCheck(hipFree(image));
	CImg<T> cpuImg(cpuData, width, height);
	free(cpuData);
	return cpuImg;
}

template<typename T>
__global__ void convolve(T *result, T *image, long imgWidth, long imgHeight, T *filter, long filWidth, long filHeight,
		long filAnchorX, long filAnchorY) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < imgWidth && y < imgHeight) {
		T value = 0;

		for (long filX = 0; filX < filWidth; ++filX) {
			long posImgX = ((x - filAnchorX + filX) + imgWidth) % imgWidth;
			for (long filY = 0; filY < filHeight; ++filY) {
				long posImgY = ((y - filAnchorY + filY) + imgHeight) % imgHeight;

				value += image[posImgY * imgWidth + posImgX] * filter[filY * filWidth + filX];
			}
		}

		result[y * imgWidth + x] = value;
	}
}

template<typename T>
__global__ void computeGradientStrengthGPU(T *gradientStrength, T *gradientX, T *gradientY, long width, long height) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		long index = y * width + x;
		gradientStrength[index] = sqrt(pow(gradientX[index], 2) + pow(gradientY[index], 2));
	}
}

template<typename T>
T * computeGradientStrength(T *grayValueImage, long width, long height) {
	T cpuSobelX[] = { 1, 0, -1, 2, 0, -2, 1, 0, -1 };
	T cpuSobelY[] = { 1, 2, 1, 0, 0, 0, -1, -2, -1 };
	T *sobelX;
	T *sobelY;
	assertCheck(hipMalloc(&sobelX, 9 * sizeof(T)));
	assertCheck(hipMalloc(&sobelY, 9 * sizeof(T)));
	assertCheck(hipMemcpy(sobelX, cpuSobelX, 9 * sizeof(T), hipMemcpyHostToDevice));
	assertCheck(hipMemcpy(sobelY, cpuSobelY, 9 * sizeof(T), hipMemcpyHostToDevice));

	T *gradientX;
	T *gradientY;
	assertCheck(hipMalloc(&gradientX, width * height * sizeof(T)));
	assertCheck(hipMalloc(&gradientY, width * height * sizeof(T)));

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
	convolve<T> <<<blocks, threads>>>(gradientX, grayValueImage, width, height, sobelX, 3, 3, 1, 1);
	assertCheck(hipGetLastError());
	convolve<T> <<<blocks, threads>>>(gradientY, grayValueImage, width, height, sobelY, 3, 3, 1, 1);
	assertCheck(hipGetLastError());

	T *gradientStrength;
	assertCheck(hipMalloc(&gradientStrength, width * height * sizeof(T)));
	computeGradientStrengthGPU<T> <<<blocks, threads>>>(gradientStrength, gradientX, gradientY, width, height);
	assertCheck(hipGetLastError());

	assertCheck(hipFree(sobelX));
	assertCheck(hipFree(sobelY));
	assertCheck(hipFree(gradientX));
	assertCheck(hipFree(gradientY));

	return gradientStrength;
}

template<typename T>
__global__ void binarizeGPU(bool *result, T *image, long width, long height, T threshold) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < width && y < height) {
		long index = y * width + x;
		if (image[index] > threshold)
			result[index] = 1;
		else
			result[index] = 0;
	}
}

//	TODO make the threshold relative to the value range within the image, instead of an absolute value
template<typename T>
bool * binarize(T *image, long width, long height, T threshold) {
	bool *binaryImage;
	assertCheck(hipMalloc(&binaryImage, width * height * sizeof(bool)));

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
	binarizeGPU<T> <<<blocks, threads>>>(binaryImage, image, width, height, threshold);
	assertCheck(hipGetLastError());

	return binaryImage;
}

template<typename T>
bool * cudaHough::preprocess(CImg<T> &image, T binarizationThreshold) {
	CImg<T> cpuGrayValueImage = RGBToGrayValueImage<T>(image);
	T *grayValueImage = cImgToGPU<T>(cpuGrayValueImage);
	T *gradientStrengthImage = computeGradientStrength<T>(grayValueImage, image.width(), image.height());
	bool *binaryImage = binarize<T>(gradientStrengthImage, image.width(), image.height(), binarizationThreshold);

	assertCheck(hipFree(grayValueImage));
	assertCheck(hipFree(gradientStrengthImage));

	return binaryImage;
}

template<typename Taccu, typename Tparam>
__global__ void computeAccumulatorArrayGPU(bool *binaryImage, long width, long height, long borderExclude,
		Taccu *accumulatorArray, Tparam minTheta, Tparam maxTheta, Tparam thetaStepSize, Tparam stepsPerRadian,
		Tparam minR, Tparam stepsPerPixel, long dimTheta) {
	long x = blockIdx.x * blockDim.x + threadIdx.x;
	long y = blockIdx.y * blockDim.y + threadIdx.y;
//	TODO calculate x and y by directly taking into account border exclude, instead of checking it afterwards
	if (x >= borderExclude && y >= borderExclude && x < width - borderExclude && y < height - borderExclude) {
		if (binaryImage[y * width + x] == 1) {
			for (Tparam theta = minTheta; theta <= maxTheta; theta += thetaStepSize) {
				Tparam r = x * cos(theta) + y * sin(theta);

				long thetaIdx = long((theta - minTheta) * stepsPerRadian);
				long rIdx = long((r - minR) * stepsPerPixel);
				accumulatorArray[rIdx * dimTheta + thetaIdx] += 1;
			}
		}
	}
}

template<typename retT, typename paramT>
retT * cudaHough::transform(bool *binaryImage, long width, long height, HoughParameterSet<paramT> &hps) {
	long dimTheta = (hps.maxTheta - hps.minTheta) * hps.stepsPerRadian + 1;
	long dimR = (hps.maxR - hps.minR) * hps.stepsPerPixel + 1;
	long borderExclude = 5;
	paramT thetaStepSize = 1.0 / hps.stepsPerRadian;

	retT *accumulatorArray;
	assertCheck(hipMalloc(&accumulatorArray, dimTheta * dimR * sizeof(retT)));
	hipMemset(accumulatorArray, 0, dimTheta * dimR * sizeof(retT));

	dim3 threads(16, 16);
	dim3 blocks((width + threads.x - 1) / threads.x, (height + threads.y - 1) / threads.y);
	computeAccumulatorArrayGPU<retT, paramT> <<<blocks, threads>>>(binaryImage, width, height, borderExclude,
			accumulatorArray, hps.minTheta, hps.maxTheta, thetaStepSize, hps.stepsPerRadian, hps.minR,
			hps.stepsPerPixel, dimTheta);
	assertCheck(hipGetLastError());

	return accumulatorArray;
}

template<typename retT, typename paramT>
std::vector<std::pair<retT, retT> > cudaHough::extractMostLikelyLines(CImg<paramT> &accumulatorArray,
		long linesToExtract) {
	return std::vector<std::pair<retT, retT> >(); // TODO return something for real
}

// Instantiate template methods so they are available to the compiler
template CImg<bool> gpuToCImg(bool *image, long width, long height);
template CImg<long> gpuToCImg(long *image, long width, long height);
template CImg<float> gpuToCImg(float *image, long width, long height);
template CImg<double> gpuToCImg(double *image, long width, long height);
template bool * cudaHough::preprocess<float>(CImg<float> &image, float binarizationThreshold);
template bool * cudaHough::preprocess<double>(CImg<double> &image, double binarizationThreshold);
template long * cudaHough::transform(bool *binaryImage, long width, long height, HoughParameterSet<float> &hps);
template long * cudaHough::transform(bool *binaryImage, long width, long height, HoughParameterSet<double> &hps);
template std::vector<std::pair<float, float> > cudaHough::extractMostLikelyLines(CImg<long> &accumulatorArray,
		long linesToExtract);
template std::vector<std::pair<double, double> > cudaHough::extractMostLikelyLines(CImg<long> &accumulatorArray,
		long linesToExtract);
